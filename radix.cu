/**
* implementation of parallel Radix Sort in CUDA using the CUB library
*
* this code compares the sorting performance between:
* - radix Sort on the GPU (via CUB DeviceRadixSort)
* - std::sort on the CPU (standard C++ sorting algorithm)
*
* the program generates a vector of 1 million random integers and measures the time
* required to sort using both methods. After sorting,
* the correctness of the result is verified and performance metrics are presented
* including the speedup obtained by the GPU version.
*
*
* result  - GPU raidx: 0.000901536 s
*         - CPU std::sort: 0.220703 s
*         - speedup: 244.808×
*
* note: this code was tested on an NVIDIA GeForce GTX 1650 GPU.
* */

#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

void checkCuda(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

bool isSorted(const std::vector<int>& v) {
    for (size_t i = 1; i < v.size(); ++i)
        if (v[i-1] > v[i]) return false;
    return true;
}

int main() {
    const size_t N = 1 << 20;
    std::vector<int> h_data(N), h_cpu(N);
    srand(42);
    for (size_t i = 0; i < N; ++i) {
        h_data[i] = rand() % 1000000;
        h_cpu[i]  = h_data[i];
    }

    int *d_keys_in, *d_keys_out;
    size_t temp_bytes = 0;
    void *d_temp = nullptr;

    checkCuda(hipMalloc(&d_keys_in,  N * sizeof(int)));
    checkCuda(hipMalloc(&d_keys_out, N * sizeof(int)));
    checkCuda(hipMemcpy(d_keys_in, h_data.data(), N * sizeof(int), hipMemcpyHostToDevice));

    hipcub::DeviceRadixSort::SortKeys(d_temp, temp_bytes,
                                   d_keys_in, d_keys_out, N);
    checkCuda(hipMalloc(&d_temp, temp_bytes));

    hipEvent_t start, stop;
    float gpuTime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipcub::DeviceRadixSort::SortKeys(d_temp, temp_bytes,
                                   d_keys_in, d_keys_out, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    checkCuda(hipMemcpy(h_data.data(), d_keys_out, N * sizeof(int), hipMemcpyDeviceToHost));

    std::cout << "GPU radix sort: " << gpuTime / 1000.0 << " s\n";

    if (!isSorted(h_data)) {
        std::cerr << "Erro: ordenação na GPU incorreta!\n";
        return 1;
    }

    float cpuTime = 0;
    hipEventRecord(start, 0);
    std::sort(h_cpu.begin(), h_cpu.end());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpuTime, start, stop);

    std::cout << "CPU std::sort:  " << cpuTime / 1000.0 << " s\n";
    std::cout << "Speedup:        " << (cpuTime / gpuTime) << "×\n";

    std::cout << "Nota: execução realizada em uma NVIDIA GeForce GTX 1650 (humilde, porém valente!)\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_keys_in);
    hipFree(d_keys_out);
    hipFree(d_temp);
    
    return 0;
}
